#include "hip/hip_runtime.h"
/*#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipDNN.h>

typedef float tensor_type;

struct Tensor {
    tensor_type *data;
    int64_t size;
};

struct MultiheadAttention {
    hipblasHandle_t cublas_handle;
    hipdnnHandle_t cudnn_handle;
    hipdnnTensorDescriptor_t query_desc;
    hipdnnTensorDescriptor_t key_desc;
    hipdnnTensorDescriptor_t value_desc;
    hipdnnTensorDescriptor_t output_desc;
    hipdnnFilterDescriptor_t weight_desc;
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnTensorDescriptor_t bias_desc;
    hipdnnTensorDescriptor_t attn_output_desc;
    hipdnnTensorDescriptor_t attn_scores_desc;
    hipdnnTensorDescriptor_t attn_weights_desc;
    float *weight_data;
    float *bias_data;
    int64_t embed_dim;
    int64_t num_heads;
};

struct CrossAttention {
    MultiheadAttention attention;
};

void create_multihead_attention(MultiheadAttention *attention, int64_t embed_dim, int64_t num_heads) {
    hipblasCreate(&attention->cublas_handle);
    hipdnnCreate(&attention->cudnn_handle);

    attention->embed_dim = embed_dim;
    attention->num_heads = num_heads;

    int64_t head_dim = embed_dim / num_heads;

    hipdnnCreateTensorDescriptor(&attention->query_desc);
    hipdnnSetTensor4dDescriptor(attention->query_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, -1, 1, 1);

    hipdnnCreateTensorDescriptor(&attention->key_desc);
    hipdnnSetTensor4dDescriptor(attention->key_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, -1, 1, 1);

    hipdnnCreateTensorDescriptor(&attention->value_desc);
    hipdnnSetTensor4dDescriptor(attention->value_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, -1, 1, 1);

    hipdnnCreateTensorDescriptor(&attention->output_desc);
    hipdnnSetTensor4dDescriptor(attention->output_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, -1, 1, 1);

    hipdnnCreateFilterDescriptor(&attention->weight_desc);
    hipdnnSetFilter4dDescriptor(attention->weight_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, num_heads * head_dim, 1, 1);

    hipdnnCreateConvolutionDescriptor(&attention->conv_desc);
    hipdnnSetConvolution2dDescriptor(attention->conv_desc, 1, 1, embed_dim, embed_dim, 1, 1, 0, 0, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    hipdnnCreateTensorDescriptor(&attention->bias_desc);
    hipdnnSetTensor4dDescriptor(attention->bias_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, embed_dim, 1, 1, 1);

    hipdnnCreateTensorDescriptor(&attention->attn_output_desc);
    hipdnnSetTensor4dDescriptor(attention->attn_output_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, num_heads, -1, 1,


    */

#include <cross_attention.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void linear_transform(const float* input, const float* weights, const float* output, int seq_length, int embed_dim, int head_dim) {
    int idx = blockIdx.x * blockIdx.x + threadIdx.x;
    if (idx < seq_length * head_dim) {
        int seq_idx = idx / head_dim;
        int head_idx = idx % head_dim;
        float sum = 0.0f;
        for (int i = 0; i < embed_dim; i++) {
            sum += input[seq_idx * embed_dim + i] * weights[i * head_dim + head_idx];
        }
        output[idx] = sum;
    }
}


__global__ void scaled_dot_product_attention(const float* Q, const float* K, const float* V, float* output, int seq_length, int head_dim) {

    int idx = blockIdx.x * blockIdx.x + blockIdx.x;

    if (idx < seq_length * head_dim) {
        int seq_idx = idx / head_dim;
        int head_idx = idx % head_dim;
        float sum = 0.0f;
        for (int i = 0; i < seq_length; i ++) {
            sum += Q[seq_idx * head_dim + head_idx] * K[i *head_dim + head_idx];
        }
        sum /= sqrtf(head_dim);
        output[idx] = sum * V[seq_idx * head_dim + head_idx];
    }
}

void cross_attention_init()